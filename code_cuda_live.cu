#include "hip/hip_runtime.h"
#include "ascon.h"
#include <cstring>
#include <iostream>
#include <fstream>
#include <vector>
#include <iomanip>
#include <sstream>
#include <random>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

// Define RATE based on the variant
#ifdef ASCON_AEAD_RATE
#define RATE ASCON_AEAD_RATE
#else
#define RATE 8
#endif

using namespace cv;
using namespace std;
namespace fs = std::filesystem;

// Helper functions
__device__ void ascon_permutation(ascon_state_t *s, int rounds);

// Ascon permutation function
__device__ void ascon_permutation(ascon_state_t *s, int rounds)
{
    static const uint8_t RC[12] = {0x0f, 0x1e, 0x2d, 0x3c, 0x4b, 0x5a, 0x69, 0x78, 0x87, 0x96, 0xa5, 0xb4};
    for (int r = 12 - rounds; r < 12; ++r)
    {
        s->x[2] ^= RC[r];
        s->x[0] ^= s->x[4];
        s->x[4] ^= s->x[3];
        s->x[2] ^= s->x[1];
        uint64_t T[5];
        for (int i = 0; i < 5; ++i)
        {
            T[i] = s->x[i] ^ (~s->x[(i + 1) % 5] & s->x[(i + 2) % 5]);
        }
        for (int i = 0; i < 5; ++i)
        {
            s->x[i] = T[i];
        }
        s->x[0] ^= s->x[4];
        s->x[4] ^= s->x[3];
        s->x[2] ^= s->x[1];
        s->x[0] = (s->x[0] >> 19) ^ (s->x[0] << (64 - 19)) ^ (s->x[0] >> 28) ^ (s->x[0] << (64 - 28));
        s->x[1] = (s->x[1] >> 61) ^ (s->x[1] << (64 - 61)) ^ (s->x[1] >> 39) ^ (s->x[1] << (64 - 39));
        s->x[2] = (s->x[2] >> 1) ^ (s->x[2] << (64 - 1)) ^ (s->x[2] >> 6) ^ (s->x[2] << (64 - 6));
        s->x[3] = (s->x[3] >> 10) ^ (s->x[3] << (64 - 10)) ^ (s->x[3] >> 17) ^ (s->x[3] << (64 - 17));
        s->x[4] = (s->x[4] >> 7) ^ (s->x[4] << (64 - 7)) ^ (s->x[4] >> 41) ^ (s->x[4] << (64 - 41));
    }
}

// AEAD functions
__device__ void ascon_loadkey(ascon_key_t *key, const uint8_t *k)
{
    std::memcpy(key->b, k, CRYPTO_KEYBYTES);
}

__device__ void ascon_initaead(ascon_state_t *s, const ascon_key_t *key, const uint8_t *npub)
{
    std::memset(s, 0, sizeof(ascon_state_t));
    s->x[0] = 0x80400c0600000000ULL ^ ((uint64_t)CRYPTO_KEYBYTES << 56) ^ ((uint64_t)ASCON_AEAD_RATE << 48);
    s->x[1] = key->x[0];
    s->x[2] = key->x[1];
    s->x[3] = ((uint64_t *)npub)[0];
    s->x[4] = ((uint64_t *)npub)[1];
    ascon_permutation(s, 12);
    s->x[3] ^= key->x[0];
    s->x[4] ^= key->x[1];
}

__device__ void ascon_adata(ascon_state_t *s, const uint8_t *ad, uint64_t adlen)
{
    while (adlen >= ASCON_AEAD_RATE)
    {
        s->x[0] ^= ((uint64_t *)ad)[0];
        ascon_permutation(s, 6);
        ad += ASCON_AEAD_RATE;
        adlen -= ASCON_AEAD_RATE;
    }
    uint8_t lastblock[ASCON_AEAD_RATE] = {0};
    std::memcpy(lastblock, ad, adlen);
    lastblock[adlen] = 0x80;
    s->x[0] ^= ((uint64_t *)lastblock)[0];
    ascon_permutation(s, 6);
    s->x[4] ^= 1;
}

__device__ void ascon_encrypt(ascon_state_t *s, uint8_t *c, const uint8_t *m, uint64_t mlen)
{
    while (mlen >= ASCON_AEAD_RATE)
    {
        s->x[0] ^= ((uint64_t *)m)[0];
        ((uint64_t *)c)[0] = s->x[0];
        ascon_permutation(s, 6);
        m += ASCON_AEAD_RATE;
        c += ASCON_AEAD_RATE;
        mlen -= ASCON_AEAD_RATE;
    }
    uint8_t lastblock[ASCON_AEAD_RATE] = {0};
    std::memcpy(lastblock, m, mlen);
    lastblock[mlen] = 0x80;
    s->x[0] ^= ((uint64_t *)lastblock)[0];
    std::memcpy(c, &s->x[0], mlen);
}

__device__ void ascon_decrypt(ascon_state_t *s, uint8_t *m, const uint8_t *c, uint64_t clen)
{
    while (clen >= ASCON_AEAD_RATE)
    {
        uint64_t cblock = ((uint64_t *)c)[0];
        ((uint64_t *)m)[0] = s->x[0] ^ cblock;
        s->x[0] = cblock;
        ascon_permutation(s, 6);
        c += ASCON_AEAD_RATE;
        m += ASCON_AEAD_RATE;
        clen -= ASCON_AEAD_RATE;
    }
    uint8_t lastblock[ASCON_AEAD_RATE] = {0};
    std::memcpy(lastblock, c, clen);
    lastblock[clen] = 0x80;
    uint64_t cblock = ((uint64_t *)lastblock)[0];
    ((uint64_t *)m)[0] = s->x[0] ^ cblock;
    s->x[0] = cblock;
}

__device__ void ascon_final(ascon_state_t *s, const ascon_key_t *k)
{
    s->x[1] ^= k->x[0];
    s->x[2] ^= k->x[1];
    ascon_permutation(s, 12);
    s->x[3] ^= k->x[0];
    s->x[4] ^= k->x[1];
}

__global__ void ascon_aead_encrypt_kernel(uint8_t *t, uint8_t *c, const uint8_t *m, uint64_t mlen, const uint8_t *ad, uint64_t adlen, const uint8_t *npub, const uint8_t *k)
{
    ascon_state_t s;
    ascon_key_t key;
    ascon_loadkey(&key, k);
    ascon_initaead(&s, &key, npub);
    ascon_adata(&s, ad, adlen);
    ascon_encrypt(&s, c, m, mlen);
    ascon_final(&s, &key);
    std::memcpy(t, &s.x[3], 16);
}

__global__ void ascon_aead_decrypt_kernel(uint8_t *m, const uint8_t *t, const uint8_t *c, uint64_t clen, const uint8_t *ad, uint64_t adlen, const uint8_t *npub, const uint8_t *k, int *result)
{
    ascon_state_t s;
    ascon_key_t key;
    ascon_loadkey(&key, k);
    ascon_initaead(&s, &key, npub);
    ascon_adata(&s, ad, adlen);
    ascon_decrypt(&s, m, c, clen - 16);
    ascon_final(&s, &key);
    *result = std::memcmp(t, &s.x[3], 16) == 0 ? 0 : -1;
}

// ...existing code...

int main() {
    cv::VideoCapture videoFace_data(0); // Use 0 for the default webcam

    if (!videoFace_data.isOpened()) {
        std::cerr << "Unable to open webcam" << std::endl;
        return -1;
    }

    int fps = 24;
    int frame_delay = 1000 / fps;

    cv::CascadeClassifier face_cascade;
    if (!face_cascade.load(cv::samples::findFile("haarcascade_frontalface_alt2.xml"))) {
        std::cerr << "Error loading face cascade" << std::endl;
        return -1;
    }

    cv::Size fixed_size(200, 200); // Set a fixed size for the ROI
    int frame_count = 0;
    int no_face_count = 0; // Counter for frames with no face detected
    const int max_no_face_frames = 30; // Maximum number of consecutive frames with no face before stopping

    std::ofstream output_file("face_detection.txt");
    if (!output_file.is_open()) {
        std::cerr << "Unable to open file for writing" << std::endl;
        return -1;
    }

    std::vector<uint8_t> key(16);
    std::vector<uint8_t> nonce(16);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);
    for (auto &byte : key)
        byte = dis(gen);
    for (auto &byte : nonce)
        byte = dis(gen);

    // Output folder path
    std::string output_folder = "E:\\ASCON\\Flow\\FLow_ascon\\image";

    // Initialize video writers
    cv::VideoWriter dot_video_writer(output_folder + "\\dot_video.mp4", cv::VideoWriter::fourcc('m', 'p', '4', 'v'), fps, fixed_size);
    cv::VideoWriter face_video_writer(output_folder + "\\face_video.mp4", cv::VideoWriter::fourcc('m', 'p', '4', 'v'), fps, cv::Size((int)videoFace_data.get(cv::CAP_PROP_FRAME_WIDTH), (int)videoFace_data.get(cv::CAP_PROP_FRAME_HEIGHT)));

    if (!dot_video_writer.isOpened() || !face_video_writer.isOpened()) {
        std::cerr << "Error opening video writers" << std::endl;
        return -1;
    }

    auto total_start = std::chrono::steady_clock::now();

    while (videoFace_data.isOpened()) {
        auto frame_start = std::chrono::steady_clock::now();

        cv::Mat frame;
        videoFace_data >> frame;
        if (frame.empty()) {
            break;
        }

        // Write the original frame to the face video
        face_video_writer.write(frame);

        // Define a fixed ROI in the center of the frame
        int x = (frame.cols - fixed_size.width) / 2;
        int y = (frame.rows - fixed_size.height) / 2;
        cv::Rect roi(x, y, fixed_size.width, fixed_size.height);

        // Crop the frame to the fixed ROI
        cv::Mat face_crop = frame(roi);

        std::vector<cv::Rect> faces;
        face_cascade.detectMultiScale(face_crop, faces, 1.1, 5, 0, cv::Size(30, 30));

        if (faces.empty()) {
            no_face_count++;
            if (no_face_count >= max_no_face_frames) {
                std::cout << "No face detected for " << max_no_face_frames << " consecutive frames. Stopping video." << std::endl;
                break;
            }
        } else {
            no_face_count = 0; // Reset the counter if a face is detected

            // Convert face_crop to a byte array
            std::vector<uint8_t> plaintext(face_crop.total() * face_crop.elemSize());
            std::memcpy(plaintext.data(), face_crop.data, plaintext.size());

            size_t plaintext_len = plaintext.size();
            std::vector<uint8_t> ciphertext(plaintext_len + 16);
            std::vector<uint8_t> tag(16);
            std::vector<uint8_t> decrypted(plaintext_len);

            uint8_t *d_plaintext, *d_ciphertext, *d_tag, *d_nonce, *d_key;
            int *d_result;
            hipMalloc(&d_plaintext, plaintext.size());
            hipMalloc(&d_ciphertext, ciphertext.size());
            hipMalloc(&d_tag, tag.size());
            hipMalloc(&d_nonce, nonce.size());
            hipMalloc(&d_key, key.size());
            hipMalloc(&d_result, sizeof(int));

            hipMemcpy(d_plaintext, plaintext.data(), plaintext.size(), hipMemcpyHostToDevice);
            hipMemcpy(d_nonce, nonce.data(), nonce.size(), hipMemcpyHostToDevice);
            hipMemcpy(d_key, key.data(), key.size(), hipMemcpyHostToDevice);

            ascon_aead_encrypt_kernel<<<1, 1>>>(d_tag, d_ciphertext, d_plaintext, plaintext_len, nullptr, 0, d_nonce, d_key);
            hipMemcpy(tag.data(), d_tag, tag.size(), hipMemcpyDeviceToHost);
            hipMemcpy(ciphertext.data(), d_ciphertext, ciphertext.size(), hipMemcpyDeviceToHost);

            ascon_aead_decrypt_kernel<<<1, 1>>>(d_plaintext, d_tag, d_ciphertext, plaintext_len + 16, nullptr, 0, d_nonce, d_key, d_result);
            hipMemcpy(decrypted.data(), d_plaintext, decrypted.size(), hipMemcpyDeviceToHost);
            int result;
            hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

            output_file << "Frame " << frame_count << std::endl;
            // print_hex(output_file, "key", key.data(), key.size());
            // print_hex(output_file, "nonce", nonce.data(), nonce.size());
            // print_hex(output_file, "plaintext", plaintext.data(), plaintext_len);
            // print_hex(output_file, "ciphertext", ciphertext.data(), plaintext_len);
            // print_hex(output_file, "tag", tag.data(), tag.size());
            // print_hex(output_file, "received", decrypted.data(), plaintext_len);

            cv::Mat color_dot_image(fixed_size.height, fixed_size.width, CV_8UC3);
            for (int i = 0; i < fixed_size.height; ++i) {
                for (int j = 0; j < fixed_size.width; ++j) {
                    int index = (i * fixed_size.width + j) * 3;
                    if (index + 2 < ciphertext.size()) {
                        color_dot_image.at<cv::Vec3b>(i, j) = cv::Vec3b(ciphertext[index], ciphertext[index + 1], ciphertext[index + 2]);
                    } else {
                        color_dot_image.at<cv::Vec3b>(i, j) = cv::Vec3b(0, 0, 0); // Padding with black if out of bounds
                    }
                }
            }
            dot_video_writer.write(color_dot_image);

            // Display the dot video
            cv::imshow("Color Dot Image", color_dot_image);

            frame_count++;

            if (cv::waitKey(frame_delay) & 0xFF == 'q') {
                break;
            }

            hipFree(d_plaintext);
            hipFree(d_ciphertext);
            hipFree(d_tag);
            hipFree(d_nonce);
            hipFree(d_key);
            hipFree(d_result);
        }

        auto frame_end = std::chrono::steady_clock::now();
        std::chrono::duration<double> frame_elapsed = frame_end - frame_start;
        output_file << "Total processing time: " << frame_elapsed.count() << " seconds" << std::endl;
    }
    videoFace_data.release();
    dot_video_writer.release();
    face_video_writer.release();
    cv::destroyAllWindows();

    output_file.close();
    std::cout << "Hex strings saved to face_detection.txt" << std::endl;

    return 0;
}
