#include "hip/hip_runtime.h"
#include "ascon.h"
#include <cstring>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>

// Define RATE based on the variant
#ifdef ASCON_AEAD_RATE
#define RATE ASCON_AEAD_RATE
#else
#define RATE 8
#endif

// Helper functions
__host__ __device__ void ascon_permutation(ascon_state_t *s, int rounds);

// Ascon permutation function
__host__ __device__ void ascon_permutation(ascon_state_t *s, int rounds)
{
    static const uint8_t RC[12] = {0x0f, 0x1e, 0x2d, 0x3c, 0x4b, 0x5a, 0x69, 0x78, 0x87, 0x96, 0xa5, 0xb4};
    for (int r = 12 - rounds; r < 12; ++r)
    {
        s->x[2] ^= RC[r];
        s->x[0] ^= s->x[4];
        s->x[4] ^= s->x[3];
        s->x[2] ^= s->x[1];
        uint64_t T[5];
        for (int i = 0; i < 5; ++i)
        {
            T[i] = s->x[i] ^ (~s->x[(i + 1) % 5] & s->x[(i + 2) % 5]);
        }
        for (int i = 0; i < 5; ++i)
        {
            s->x[i] = T[i];
        }
        s->x[0] ^= s->x[4];
        s->x[4] ^= s->x[3];
        s->x[2] ^= s->x[1];
        s->x[0] = (s->x[0] >> 19) ^ (s->x[0] << (64 - 19)) ^ (s->x[0] >> 28) ^ (s->x[0] << (64 - 28));
        s->x[1] = (s->x[1] >> 61) ^ (s->x[1] << (64 - 61)) ^ (s->x[1] >> 39) ^ (s->x[1] << (64 - 39));
        s->x[2] = (s->x[2] >> 1) ^ (s->x[2] << (64 - 1)) ^ (s->x[2] >> 6) ^ (s->x[2] << (64 - 6));
        s->x[3] = (s->x[3] >> 10) ^ (s->x[3] << (64 - 10)) ^ (s->x[3] >> 17) ^ (s->x[3] << (64 - 17));
        s->x[4] = (s->x[4] >> 7) ^ (s->x[4] << (64 - 7)) ^ (s->x[4] >> 41) ^ (s->x[4] << (64 - 41));
    }
}

// AEAD functions
__host__ __device__ void ascon_loadkey(ascon_key_t *key, const uint8_t *k)
{
    memcpy(key->b, k, CRYPTO_KEYBYTES);
}

__host__ __device__ void ascon_initaead(ascon_state_t *s, const ascon_key_t *key, const uint8_t *npub)
{
    memset(s, 0, sizeof(ascon_state_t));
    s->x[0] = 0x80400c0600000000ULL ^ ((uint64_t)CRYPTO_KEYBYTES << 56) ^ ((uint64_t)ASCON_AEAD_RATE << 48);
    s->x[1] = key->x[0];
    s->x[2] = key->x[1];
    s->x[3] = ((uint64_t *)npub)[0];
    s->x[4] = ((uint64_t *)npub)[1];
    ascon_permutation(s, 12);
    s->x[3] ^= key->x[0];
    s->x[4] ^= key->x[1];
}

__host__ __device__ void ascon_adata(ascon_state_t *s, const uint8_t *ad, uint64_t adlen)
{
    while (adlen >= ASCON_AEAD_RATE)
    {
        s->x[0] ^= ((uint64_t *)ad)[0];
        ascon_permutation(s, 6);
        ad += ASCON_AEAD_RATE;
        adlen -= ASCON_AEAD_RATE;
    }
    uint8_t lastblock[ASCON_AEAD_RATE] = {0};
    memcpy(lastblock, ad, adlen);
    lastblock[adlen] = 0x80;
    s->x[0] ^= ((uint64_t *)lastblock)[0];
    ascon_permutation(s, 6);
    s->x[4] ^= 1;
}

__host__ __device__ void ascon_decrypt(ascon_state_t *s, uint8_t *m, const uint8_t *c, uint64_t clen)
{
    while (clen >= ASCON_AEAD_RATE)
    {
        uint64_t cblock = ((uint64_t *)c)[0];
        ((uint64_t *)m)[0] = s->x[0] ^ cblock;
        s->x[0] = cblock;
        ascon_permutation(s, 6);
        c += ASCON_AEAD_RATE;
        m += ASCON_AEAD_RATE;
        clen -= ASCON_AEAD_RATE;
    }
    uint8_t lastblock[ASCON_AEAD_RATE] = {0};
    memcpy(lastblock, c, clen);
    lastblock[clen] = 0x80;
    uint64_t cblock = ((uint64_t *)lastblock)[0];
    ((uint64_t *)m)[0] = s->x[0] ^ cblock;
    s->x[0] = cblock;
}

__host__ __device__ void ascon_final(ascon_state_t *s, const ascon_key_t *k)
{
    s->x[1] ^= k->x[0];
    s->x[2] ^= k->x[1];
    ascon_permutation(s, 12);
    s->x[3] ^= k->x[0];
    s->x[4] ^= k->x[1];
}

__device__ int ascon_compare(const uint8_t *a, const uint8_t *b, size_t len)
{
    for (size_t i = 0; i < len; ++i)
    {
        if (a[i] != b[i])
        {
            return -1;
        }
    }
    return 0;
}

__global__ void ascon_aead_decrypt_kernel(uint8_t *m, const uint8_t *tag, const uint8_t *c, uint64_t clen, const uint8_t *ad, uint64_t adlen, const uint8_t *npub, const uint8_t *k, int *result)
{
    ascon_state_t s;
    ascon_key_t key;
    ascon_loadkey(&key, k);
    ascon_initaead(&s, &key, npub);
    if (adlen > 0)
    {
        ascon_adata(&s, ad, adlen);
    }
    ascon_decrypt(&s, m, c, clen - CRYPTO_ABYTES);
    s.x[0] ^= 0x80;
    ascon_final(&s, &key);
    *result = ascon_compare(tag, (uint8_t *)&s.x[3], CRYPTO_ABYTES);
}

int main()
{
    std::vector<uint8_t> key(16);
    std::vector<uint8_t> nonce(16);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);
    for (auto &byte : key)
        byte = dis(gen);
    for (auto &byte : nonce)
        byte = dis(gen);

    std::vector<uint8_t> ciphertext = { /* your ciphertext data */ };
    std::vector<uint8_t> tag = { /* your tag data */ };
    size_t ciphertext_len = ciphertext.size();
    std::vector<uint8_t> decrypted(ciphertext_len - 16);

    uint8_t *d_plaintext, *d_ciphertext, *d_tag, *d_nonce, *d_key;
    int *d_result;
    hipMalloc(&d_plaintext, decrypted.size());
    hipMalloc(&d_ciphertext, ciphertext.size());
    hipMalloc(&d_tag, tag.size());
    hipMalloc(&d_nonce, nonce.size());
    hipMalloc(&d_key, key.size());
    hipMalloc(&d_result, sizeof(int));

    hipMemcpy(d_ciphertext, ciphertext.data(), ciphertext.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_tag, tag.data(), tag.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_nonce, nonce.data(), nonce.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key.data(), key.size(), hipMemcpyHostToDevice);

    auto start_decrypt = std::chrono::high_resolution_clock::now();
    ascon_aead_decrypt_kernel<<<1, 1>>>(d_plaintext, d_tag, d_ciphertext, ciphertext_len, nullptr, 0, d_nonce, d_key, d_result);
    hipDeviceSynchronize();
    auto end_decrypt = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_decrypt = end_decrypt - start_decrypt;
    std::cout << "Decryption time: " << elapsed_decrypt.count() << " seconds" << std::endl;

    hipMemcpy(decrypted.data(), d_plaintext, decrypted.size(), hipMemcpyDeviceToHost);
    int result;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyHostToDevice);

    if (result == 0) {
        std::cout << "Decryption successful" << std::endl;
    } else {
        std::cout << "Decryption failed" << std::endl;
    }

    // Print or save the decrypted data as needed

    hipFree(d_plaintext);
    hipFree(d_ciphertext);
    hipFree(d_tag);
    hipFree(d_nonce);
    hipFree(d_key);
    hipFree(d_result);

    return 0;
}
